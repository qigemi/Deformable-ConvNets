/*!
 * Copyright (c) 2017 Microsoft
 * Licensed under The Apache-2.0 License [see LICENSE for details]
 * \file deformable_convolution.cu
 * \brief
 * \author Yuwen Xiong, Haozhi Qi, Jifeng Dai
*/

#include "./bilinear-inl.h"
#include <vector>

namespace mxnet {
  namespace op {

    template<>
    Operator* CreateOp<gpu>(BilinearParam param, int dtype,
      std::vector<TShape> *in_shape,
      std::vector<TShape> *out_shape,
      Context ctx) {
      Operator *op = NULL;
      MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
        op = new BilinearOp<gpu, DType>(param);
      })
        return op;
    }

  }  // namespace op
}  // namespace mxnet

